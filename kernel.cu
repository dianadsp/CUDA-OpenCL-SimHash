
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <iterator>
#include <unordered_map>
#include <bitset>
#include <thread>
#include <math.h> 

#define BITS 64
#define MAX_LEN_WORD 20

using namespace std;

string readFile(string file) {
    ifstream MyFile;
    MyFile.open(file);
    stringstream strStream;
    strStream << MyFile.rdbuf();
    return strStream.str();
}

vector<string> strToVector(string data, char stop_w = '\n') {
    vector<string> out;
    size_t last_init = 0;
    for (size_t i = 0; i < data.length(); i++) {
        if (data[i] == stop_w) {
            string temp = data.substr(last_init, i - last_init);
            out.push_back(temp);
            last_init = i + 1;
        }
    }
    return out;
}

void printVector(vector<string> data) {
    for (size_t i = 0; i < data.size(); i++) {
        printf("%s\n", data[i].c_str());
    }
}

void printVectorInt(vector<int> data) {
    for (size_t i = 0; i < data.size(); i++) {
        printf("%i ", data[i]);
    }
    printf("\n");
}

void printMap(unordered_map <string, int> data) {
    for (unordered_map<string, int >::const_iterator it = data.begin(); it != data.end(); ++it) {
        cout << it->first << " " << it->second << "\n";
    }
}

string* text_generator(vector<string> my_dict, int len_words) {
    string* text = new string[len_words];
    for (size_t i = 0; i < len_words; i++) {
        text[i] = my_dict[rand() % my_dict.size()];
    }
    return text;
}

unordered_map <string, int> get_frec(string* words, int len_words) {

    unordered_map <string, int> out;
    for (size_t i = 0; i < len_words; i++) {
        if (out.count(words[i]) > 0) { // cuantas veces parece la palabra en el texto
            out[words[i]] += 1;
        }
        else {
            out[words[i]] = 1;
        }
    }
    return out;
}

string sum_matrix(vector<long*> in_matrix) {
    string out;
    for (size_t j = 0; j < BITS; j++) {
        long int temp = 0;
        for (size_t i = 0; i < in_matrix.size(); i++) {
            temp += in_matrix[i][j];
        }
        if (temp > 0) {
            out += '1';
        }
        else {
            out += '0';
        }
    }
    return out;
}

size_t count_words(unordered_map <string, int>* in_words, int n_text) {
    size_t count_l = 0;
    for (size_t i = 0; i < n_text; i++) {
        count_l += in_words[i].size();
    }
    return count_l;
}

char* strToChar(string data) {
    char* out = new char[MAX_LEN_WORD];
    for (size_t i = 0; i < MAX_LEN_WORD; i++) {
        if (i < data.size()) {
            out[i] = data[i];
        }
        else {
            out[i] = 0;
        }
    }
    return out;
}

void compress_sim_data_cuda(unordered_map <string, int>* in_words, char * & s_out, int * & f_out , int n_text) {

    size_t numerate = 0;
    for (size_t i = 0; i < n_text; i++) {
        for (unordered_map<string, int >::const_iterator it = in_words[i].begin(); it != in_words[i].end(); ++it) {
            char* t_str = strToChar(it->first);
            for (size_t i = 0; i < MAX_LEN_WORD; i++) {
                s_out[numerate * MAX_LEN_WORD + i] = t_str[i];
            }
            f_out[numerate] = it->second;
            numerate++;
        }
    }
}

string* extract_sim_data_cuda(long * v_words, unordered_map <string, int>* in_words, int n_text) {
    string* out = new string[n_text];
    size_t numerate = 0;
    for (size_t i = 0; i < n_text; i++) {
        vector<long*> temp;
        for (unordered_map<string, int >::const_iterator it = in_words[i].begin(); it != in_words[i].end(); ++it) {
            long * t_bits = new long[BITS];
            for (size_t b = 0; b < BITS; b++) {
                t_bits[b] = v_words[numerate + b];
            }
            temp.push_back(t_bits);
            numerate+= BITS;
        }
        out[i] = sum_matrix(temp);
    }
    return out;
}


void sim_hash_lineal(char* s_in, int* f_in, long* & out, int len) {
    for (size_t t = 0; t < len; t++) {
        unsigned long long int hash = 5381;
        for (size_t i = 0; i < MAX_LEN_WORD; i++) {
            if (s_in[t * MAX_LEN_WORD + i] != 0) {
                hash = ((hash << 5) + hash) + (int)s_in[t * MAX_LEN_WORD + i];
            }
        }
        bool* bits = new bool[BITS];
        for (size_t i = 0; i < BITS; i++) {
            bits[i] = hash % 2;
            hash = hash / 2;
        }
        for (size_t i = 0; i < BITS; i++) {
            size_t p = (t * BITS) + i;
            out[p] = (int)bits[i];
            if (out[p] == 1) {
                out[p] += f_in[t];
            }
            else {
                out[p] -= f_in[t];
            }
        }
    }
}

__global__ void cuda_sim_hash(char * s_in, int * f_in, long * out, int len)
{
    int t = (blockIdx.x * blockDim.x) + (threadIdx.x);
    if (t >= 0 && t < len) {
        unsigned long long int hash = 5381;
        for (size_t i = 0; i < MAX_LEN_WORD; i++) {
            if (s_in[t * MAX_LEN_WORD + i] != 0) {
                hash = ((hash << 5) + hash) + (int)s_in[t * MAX_LEN_WORD + i];
            }
        }
        bool* bits = new bool[BITS];
        for (size_t i = 0; i < BITS; i++) {
            bits[i] = hash % 2;
            hash = hash / 2;
        }
        for (size_t i = 0; i < BITS; i++) {
            size_t p = (t * BITS) + i;
            out[p] = (int)bits[i];
            if (out[p] == 1) {
                out[p] += f_in[t];
            }
            else {
                out[p] -= f_in[t];
            }
        }
        delete bits;
    }
}


bool compare_str(string * a, string * b, int len) {
    for (size_t i = 0; i < len; i++) {
        if (a[i] != b[i]) {
            return false;
        }
    }
    return true;
}

int main() {

    clock_t begin, end;
    double elapsed_secs;
    long long int w_s; //palabras por segundo

    srand(time(NULL));
    
    string words = readFile("words.txt"); //leer palabras
    vector<string> l_words = strToVector(words); //libreria de palabras

    int long_text = 4000; //longitud de palabras por texto
    int n_text = 2560; //cantidad de textos (documentos)

    printf("Num. textos: %i, Long Text: %i \n", n_text, long_text);

    unordered_map <string, int>* words_frec = new unordered_map <string, int>[n_text]; 
    // diccionario de frecuencias por palabra de cada documento

    for (size_t i = 0; i < n_text; i++) {
        words_frec[i] = get_frec(text_generator(l_words, long_text), long_text);
    }

    size_t amount_words = count_words(words_frec, n_text);

    printf("Total Words: %i words \n", amount_words);

    char * s_in = new char[amount_words * MAX_LEN_WORD];
    int * f_in = new int[amount_words];
    long * out = new long[amount_words * BITS];

    compress_sim_data_cuda(words_frec, s_in, f_in , n_text);

    char* cu_s_in = 0;
    int* cu_f_in = 0;
    long* cu_out = 0;

    hipMalloc((void**)&cu_s_in, amount_words * sizeof(char) * MAX_LEN_WORD);
    hipMalloc((void**)&cu_f_in, amount_words * sizeof(int));
    hipMalloc((void**)&cu_out, amount_words * sizeof(long) * BITS);

    begin = clock();

    hipMemcpy(cu_s_in, s_in, amount_words * sizeof(char) * MAX_LEN_WORD, hipMemcpyHostToDevice);
    hipMemcpy(cu_f_in, f_in, amount_words * sizeof(int), hipMemcpyHostToDevice);

    int thr = 1024;
    int dim_grid = (amount_words/thr)+1;    

    cuda_sim_hash <<< dim_grid, thr >>> (cu_s_in, cu_f_in, cu_out, amount_words);

    hipMemcpy(out, cu_out, amount_words * sizeof(long) * BITS, hipMemcpyDeviceToHost);

    end = clock();
    elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    printf("Grilla: %d, Bloque:  %d \n", dim_grid, thr);
    printf("Tiempo Cuda:  %f ms \n", elapsed_secs);
    w_s = amount_words / elapsed_secs;
    printf("palabras por Seg:  %d words \n", w_s);
    
    string* r_out = extract_sim_data_cuda(out, words_frec, n_text);



    long* out_l = new long[amount_words * BITS];

    begin = clock();

    sim_hash_lineal(s_in, f_in, out_l, amount_words);

    end = clock();
    elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    printf("Tiempo Lineal:  %f ms \n", elapsed_secs);
    w_s = amount_words / elapsed_secs;
    printf("palabras por Seg:  %d words \n", w_s);

    string* r_out_l = extract_sim_data_cuda(out_l, words_frec, n_text);

    /*
    for (size_t i = 0; i < n_text; i++) {
        cout << r_out_l[i] << endl;
    }*/

    if (compare_str(r_out, r_out_l, n_text)) {
        cout << "Ok" << endl;
    }
    else {
        cout << "Error" << endl;
    }
}

